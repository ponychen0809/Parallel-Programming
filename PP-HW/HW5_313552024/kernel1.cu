#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int maxIterations) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, 
                            int* output, int resX, int resY, int maxIterations) {
    // 計算目前執行緒處理的像素位置
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (thisX >= resX || thisY >= resY) return;
    
    // 計算複數平面上的座標
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;
    
    // 計算該點的迭代次數並儲存結果
    int index = thisY * resX + thisX;
    output[index] = mandel(x, y, maxIterations);
}

void hostFE(float upperX, float upperY, float lowerX, float lowerY, 
            int* img, int resX, int resY, int maxIterations) {
    // 計算步長
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    
    // 分配主機和設備記憶體
    int* device_output;
    int size = resX * resY * sizeof(int);
    int* host_output = (int*)malloc(size);
    
    hipMalloc(&device_output, size);
    
    // 設定執行組態
    dim3 blockDim(16, 16);  // 每個區塊16x16個執行緒
    dim3 gridDim((resX + blockDim.x - 1) / blockDim.x, 
                 (resY + blockDim.y - 1) / blockDim.y);
    
    // 啟動核心
    mandelKernel<<<gridDim, blockDim>>>(lowerX, lowerY, stepX, stepY, 
                                       device_output, resX, resY, maxIterations);
    
    // 複製結果回主機
    hipMemcpy(host_output, device_output, size, hipMemcpyDeviceToHost);
    memcpy(img, host_output, size);
    
    // 釋放記憶體
    hipFree(device_output);
    free(host_output);
} 