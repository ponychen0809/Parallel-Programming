#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// 使用常數記憶體來儲存固定的參數
__constant__ float d_lowerX, d_lowerY, d_stepX, d_stepY;

__device__ __forceinline__ int mandel(float c_re, float c_im, int maxIterations) {
    float z_re = c_re, z_im = c_im;
    float z_re2 = z_re * z_re, z_im2 = z_im * z_im;
    
    int i;
    #pragma unroll 8
    for (i = 0; i < maxIterations; ++i) {
        if (z_re2 + z_im2 > 4.f)
            break;
            
        z_im = 2.f * z_re * z_im + c_im;
        z_re = z_re2 - z_im2 + c_re;
        z_re2 = z_re * z_re;
        z_im2 = z_im * z_im;
    }
    return i;
}

__global__ void mandelKernel(int* __restrict__ output, 
                            const int resX, const int resY, 
                            const int maxIterations,
                            const size_t pitch) {
    // 使用共享記憶體來儲存中間結果
    __shared__ int cache[32][32];
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x * blockDim.x;
    const int by = blockIdx.y * blockDim.y;
    const int x = bx + tx;
    const int y = by + ty;
    
    // 每個執行緒處理多個像素點
    if (x < resX && y < resY) {
        float real = d_lowerX + x * d_stepX;
        float imag = d_lowerY + y * d_stepY;
        
        cache[ty][tx] = mandel(real, imag, maxIterations);
        
        // 使用 pitched memory 來寫入全域記憶體
        int* row = (int*)((char*)output + y * pitch);
        row[x] = cache[ty][tx];
    }
}

void hostFE(float upperX, float upperY, float lowerX, float lowerY, 
            int* img, int resX, int resY, int maxIterations) {
    // 計算步長
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    
    // 將常用參數複製到常數記憶體
    hipMemcpyToSymbol(HIP_SYMBOL(d_lowerX), &lowerX, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_lowerY), &lowerY, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_stepX), &stepX, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_stepY), &stepY, sizeof(float));
    
    // 使用 page-locked memory
    int* host_output;
    hipHostAlloc(&host_output, resX * resY * sizeof(int), hipHostMallocDefault);
    
    // 使用 pitched memory
    int* device_output;
    size_t pitch;
    hipMallocPitch(&device_output, &pitch, resX * sizeof(int), resY);

    dim3 blockDim(32, 32);
    dim3 gridDim((resX + blockDim.x - 1) / blockDim.x, 
                 (resY + blockDim.y - 1) / blockDim.y);
    
    // 使用 stream 來重疊計算和記憶體傳輸
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    mandelKernel<<<gridDim, blockDim, 0, stream>>>(
        device_output, resX, resY, maxIterations, pitch);
    
    // 使用 2D 記憶體複製
    hipMemcpy2DAsync(host_output, resX * sizeof(int),
                      device_output, pitch,
                      resX * sizeof(int), resY,
                      hipMemcpyDeviceToHost, stream);
    
    hipStreamSynchronize(stream);
    
    // 複製結果到輸出圖像
    memcpy(img, host_output, resX * resY * sizeof(int));
    

    hipStreamDestroy(stream);
    hipFree(device_output);
    hipHostFree(host_output);
}