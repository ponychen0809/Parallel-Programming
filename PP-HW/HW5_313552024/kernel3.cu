#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int maxIterations) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;
        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, 
                            int* output, int pitch, int resX, int resY, int maxIterations) {
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    int strideX = gridDim.x * blockDim.x;
    int strideY = gridDim.y * blockDim.y;
    
    for (int y = thisY; y < resY; y += strideY) {
        for (int x = thisX; x < resX; x += strideX) {
            float real = lowerX + x * stepX;
            float imag = lowerY + y * stepY;
            int* row = (int*)((char*)output + y * pitch);
            row[x] = mandel(real, imag, maxIterations);
        }
    }
}

void hostFE(float upperX, float upperY, float lowerX, float lowerY, 
            int* img, int resX, int resY, int maxIterations) {
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    
    int* device_output;
    size_t pitch;
    int* host_output;
    
    // 使用page-locked memory分配主機記憶體
    hipHostAlloc(&host_output, resX * resY * sizeof(int), hipHostMallocDefault);
    // 使用pitched memory分配設備記憶體
    hipMallocPitch(&device_output, &pitch, resX * sizeof(int), resY);
    
    dim3 blockDim(16, 16);
    dim3 gridDim((resX + blockDim.x - 1) / blockDim.x, 
                 (resY + blockDim.y - 1) / blockDim.y);
    
    // 啟動kernel
    mandelKernel<<<gridDim, blockDim>>>(lowerX, lowerY, stepX, stepY, 
                                       device_output, pitch, resX, resY, maxIterations);
    
    // 使用2D記憶體複製將結果從設備複製到主機
    hipMemcpy2D(host_output, resX * sizeof(int), device_output, pitch, 
                 resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, host_output, resX * resY * sizeof(int));
    

    hipFree(device_output);
    hipHostFree(host_output);
} 